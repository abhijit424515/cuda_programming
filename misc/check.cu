#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    std::cout << "Compute capability: " << props.major << "." << props.minor << std::endl;
    return 0;
}

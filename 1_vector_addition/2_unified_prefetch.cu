#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (tid < N) c[tid] = a[tid] + b[tid];
}

int main() {
  constexpr int N = 1 << 16;
  constexpr size_t bytes = sizeof(int) * N;

  int *a, *b, *c;

  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);
  
  // Get the device ID for prefetching calls
  int id = hipGetDevice(&id);

  // Set some hints about the data and do some prefetching
  hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, id);

  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }
  
  // Pre-fetch 'a' and 'b' arrays to the specified device (GPU)
  hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
  hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
  hipMemPrefetchAsync(a, bytes, id);
  hipMemPrefetchAsync(b, bytes, id);
  
  int BLOCK_SIZE = 1 << 10;
  int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // Call CUDA kernel
  vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);
  hipDeviceSynchronize();

  // Prefetch to the host (CPU)
  hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }
  hipFree(a);
  hipFree(b);
  hipFree(c);

  std::cout << "COMPLETED SUCCESSFULLY!\n";
  return 0;
}